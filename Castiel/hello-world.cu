// hello-world.cu 

#include <hip/hip_runtime.h>
#include <stdio.h> 
__global__ void cuda_function()
{
   printf("Hello World from GPU!\n"); 
    __syncthreads();               // to synchronize all threads
}

int main()
{
    cuda_function<<<1,1>>>();
    hipDeviceSynchronize();      // to synchronize device call
    return 0;
}